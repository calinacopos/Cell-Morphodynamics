
#ifndef _LS_DIFFCENTRAL
#define _LS_DIFFCENTRAL

#include "ls_common.cu"

__device__ void diffCentral(
	SHARED_DATA_DEF,
	float & dx, float & dy, float & dz
	)
{
	dx = (NHOOD(1,0,0) - NHOOD(-1,0,0))*0.5;
	dy = (NHOOD(0,1,0) - NHOOD(0,-1,0))*0.5;
	dz = (NHOOD(0,0,1) - NHOOD(0,0,-1))*0.5;
}

#endif
