
#ifndef _LS_COMMON
#define _LS_COMMON

#define SHARED_DATA s_Data
#define SHARED_DATA_DEF float SHARED_DATA[BLOCKDIM_Z + 2*PADDING][BLOCKDIM_Y + 2*PADDING][BLOCKDIM_X + 2*PADDING]

#define DATA SHARED_DATA[threadIdx.z][threadIdx.y][threadIdx.x]
#define NHOOD(i,j,k) SHARED_DATA[threadIdx.z+k][threadIdx.y+j][threadIdx.x+i]

#endif
