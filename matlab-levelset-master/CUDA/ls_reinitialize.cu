#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "ls_common.cu"
#include "ls_preparedata.cu"
#include "ls_sign.cu"
#include "ls_godunov.cu"

__global__ void reinitialize(
	const float * phi,
	float * out,
	const int * dim
	)
{
	__shared__ SHARED_DATA_DEF;

	int ind = 0;
	if (!prepareData(phi, SHARED_DATA, dim, ind))
		return;

	__syncthreads();

	float S = sign(SHARED_DATA);

	float dx2, dy2, dz2;
	godunov(SHARED_DATA, dx2, dy2, dz2, S);

	out[ind] = S *(1.0 - sqrtf(dx2 + dy2 + dz2));
}
