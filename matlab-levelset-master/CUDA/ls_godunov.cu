#include "hip/hip_runtime.h"

#ifndef _LS_GODONUV
#define _LS_GODONUV

#include "ls_common.cu"
#include "ls_diffonesided.cu"

__device__ void godunov(
	SHARED_DATA_DEF,
	float & dx2, float & dy2, float & dz2, float a
	)
{
	float dxm, dxp, dym, dyp, dzm, dzp;
	diffOnesided(SHARED_DATA, dxm, dxp, dym, dyp, dzm, dzp);

	if (a > 0) {
		dx2 = max( powf(max(dxm,0.0f),2), powf(min(dxp,0.0f),2) );
		dy2 = max( powf(max(dym,0.0f),2), powf(min(dyp,0.0f),2) );
		dz2 = max( powf(max(dzm,0.0f),2), powf(min(dzp,0.0f),2) );
	}
	else {
		dx2 = max( powf(min(dxm,0.0f),2), powf(max(dxp,0.0f),2) );
		dy2 = max( powf(min(dym,0.0f),2), powf(max(dyp,0.0f),2) );
		dz2 = max( powf(min(dzm,0.0f),2), powf(max(dzp,0.0f),2) );
	}
}

#endif
