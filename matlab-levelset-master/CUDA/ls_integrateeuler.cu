#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

__global__ void integrateEuler(
	float * phi,
	const float * ddt,
	const float dt,
	const int * dim
	)
{
    // Separate blockIdx.yz
    const int blockIdx_y = blockIdx.y % (dim[1] / BLOCKDIM_Y);
    const int blockIdx_z = blockIdx.y / (dim[1] / BLOCKDIM_Y);

	// Compute grid coordinates
    const int baseX = blockIdx.x * BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx_y * BLOCKDIM_Y + threadIdx.y;
    const int baseZ = blockIdx_z * BLOCKDIM_Z + threadIdx.z;

	// Compute linear index to grid point
	int ind = (baseZ*dim[1] + baseY)*dim[0] + baseX;

	// Propagate
    phi[ind] += dt*ddt[ind];
}
