
#ifndef _LS_DIFFONESIDED
#define _LS_DIFFONESIDED

#include "ls_common.cu"

__device__ void diffOnesided(
	SHARED_DATA_DEF,
	float & dxm, float & dxp,
	float & dym, float & dyp,
	float & dzm, float & dzp
	)
{
	dxm = DATA - NHOOD(-1,0,0);
	dxp = NHOOD(1,0,0) - DATA;

	dym = DATA - NHOOD(0,-1,0);
	dyp = NHOOD(0,1,0) - DATA;

	dzm = DATA - NHOOD(0,0,-1);
	dzp = NHOOD(0,0,1) - DATA;
}

#endif
