
#ifndef _LS_PREPAREDATA
#define _LS_PREPAREDATA

#include "ls_common.cu"

__device__ bool prepareData(
	const float * data,
	SHARED_DATA_DEF,
	const int * dim,
	int & ind
	)
{
    // Separate blockIdx.yz
    const int blockIdx_y = blockIdx.y % (dim[1] / BLOCKDIM_Y);
    const int blockIdx_z = blockIdx.y / (dim[1] / BLOCKDIM_Y);

	// Compute grid coordinates
    const int baseX = blockIdx.x * BLOCKDIM_X - PADDING + threadIdx.x;
    const int baseY = blockIdx_y * BLOCKDIM_Y - PADDING + threadIdx.y;
    const int baseZ = blockIdx_z * BLOCKDIM_Z - PADDING + threadIdx.z;

	// Compute linear index to grid point
	ind = (baseZ*dim[1] + baseY)*dim[0] + baseX;

	// Load the data to the shared memory
	if (baseX >= 0 && baseX < dim[0] &&
		baseY >= 0 && baseY < dim[1] &&
		baseZ >= 0 && baseZ < dim[2])
		DATA = data[ind];
	else
		DATA = 0;

	// Return false if the thread is in the padded region
	if (threadIdx.x < PADDING || threadIdx.x >= BLOCKDIM_X+PADDING ||
		threadIdx.y < PADDING || threadIdx.y >= BLOCKDIM_Y+PADDING ||
		threadIdx.z < PADDING || threadIdx.z >= BLOCKDIM_Z+PADDING)
		return false;

	return true;
}

#endif
