
#ifndef _LS_DIFF2
#define _LS_DIFF2

#include "ls_common.cu"

__device__ void diff2(
	SHARED_DATA_DEF,
	float & d2x, float & d2y, float & d2z, float & dxdy, float & dxdz, float & dydz
	)
{
	d2x = NHOOD(1,0,0) - 2.0f*DATA + NHOOD(-1,0,0);
	d2y = NHOOD(0,1,0) - 2.0f*DATA + NHOOD(0,-1,0);
	d2z = NHOOD(0,0,1) - 2.0f*DATA + NHOOD(0,0,-1);

	dxdy = (-NHOOD(1,1,0) + NHOOD(1,-1,0) - NHOOD(-1,-1,0) + NHOOD(-1,1,0)) * 0.25f;
	dxdz = (-NHOOD(1,0,1) + NHOOD(1,0,-1) - NHOOD(-1,0,-1) + NHOOD(-1,0,1)) * 0.25f;
	dydz = (-NHOOD(0,1,1) + NHOOD(0,1,-1) - NHOOD(0,-1,-1) + NHOOD(0,-1,1)) * 0.25f;
}

#endif
