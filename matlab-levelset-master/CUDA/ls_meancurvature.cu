#include "hip/hip_runtime.h"

#ifndef _LS_MEANCURVATURE
#define _LS_MEANCURVATURE

#include "ls_common.cu"
#include "ls_diffcentral.cu"
#include "ls_diff2.cu"

__device__ void meanCurvature(
	SHARED_DATA_DEF,
	float & c
	)
{
	float dx, dy, dz;
	diffCentral(SHARED_DATA, dx, dy, dz);

	float dx2 = dx*dx;
	float dy2 = dy*dy;
	float dz2 = dz*dz;
	float denominator = powf(2.0f*(dx2 + dy2 + dz2), 1.5f);

	if (denominator < 0.00035) { // Roughly sqrt(float epsilon)
		c = 0;
		return;
	}

	float d2x, d2y, d2z, dxdy, dxdz, dydz;
	diff2(SHARED_DATA, d2x, d2y, d2z, dxdy, dxdz, dydz);


	float nominator = dx2*(d2y + d2z) - 2.0f*dy*dz*dydz +
					  dy2*(d2x + d2z) - 2.0f*dx*dz*dxdz +
					  dz2*(d2x + d2y) - 2.0f*dx*dy*dxdy;
	c = nominator / denominator;
}

#endif
