#include "hip/hip_runtime.h"

#ifndef _LS_SIGN
#define _LS_SIGN

#include "ls_common.cu"
#include "ls_diffcentral.cu"

__device__ float sign(SHARED_DATA_DEF)
{
	float dx, dy, dz;
	diffCentral(SHARED_DATA, dx, dy, dz);

	return DATA / sqrtf(DATA*DATA + dx*dx + dy*dy + dz*dz);
}

#endif
