#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "ls_common.cu"
#include "ls_preparedata.cu"
#include "ls_godunov.cu"
#include "ls_meancurvature.cu"

__global__ void speedNormal(
	const float * phi,
	float * out,
	const int * dim,
	const float * F,
	const float a
	)
{
	__shared__ SHARED_DATA_DEF;

	int ind = 0;
	if (!prepareData(phi, SHARED_DATA, dim, ind))
		return;

	__syncthreads();

	float dx2, dy2, dz2;
	godunov(SHARED_DATA, dx2, dy2, dz2, F[ind]);

	float dx, dy, dz;
	diffCentral(SHARED_DATA, dx, dy, dz);

	float c;
	meanCurvature(SHARED_DATA, c);

	out[ind] = -F[ind]*sqrtf(dx2 + dy2 + dz2) + a*c*sqrtf(dx*dx + dy*dy + dz*dz);
}
